
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdint>
#include <cassert>

__host__
inline void cudaAssert(hipError_t code, const char* file, int line)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "CUDA_ASSERT: \"%s\" @ %s : %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}
#define CUDA_ASSERT(code) do { cudaAssert(code, __FILE__, __LINE__); } while(0)

typedef std::uint32_t crc32_t;
#define CRC32_MAX (UINT32_MAX)
#define CRC32_MIN (UINT32_MIN)

#define CRC32_POLYNOMIAL (0x04C11DB7)
#define CRC32_TABLE_SIZE (256)
__device__
const crc32_t crc32_table[CRC32_TABLE_SIZE] = {
    0x00000000, 0x04C11DB7, 0x09823B6E, 0x0D4326D9, 0x130476DC, 0x17C56B6B, 0x1A864DB2, 0x1E475005,  //   0 [0x00 .. 0x07]
    0x2608EDB8, 0x22C9F00F, 0x2F8AD6D6, 0x2B4BCB61, 0x350C9B64, 0x31CD86D3, 0x3C8EA00A, 0x384FBDBD,  //   8 [0x08 .. 0x0F]
    0x4C11DB70, 0x48D0C6C7, 0x4593E01E, 0x4152FDA9, 0x5F15ADAC, 0x5BD4B01B, 0x569796C2, 0x52568B75,  //  16 [0x10 .. 0x17]
    0x6A1936C8, 0x6ED82B7F, 0x639B0DA6, 0x675A1011, 0x791D4014, 0x7DDC5DA3, 0x709F7B7A, 0x745E66CD,  //  24 [0x18 .. 0x1F]
    0x9823B6E0, 0x9CE2AB57, 0x91A18D8E, 0x95609039, 0x8B27C03C, 0x8FE6DD8B, 0x82A5FB52, 0x8664E6E5,  //  32 [0x20 .. 0x27]
    0xBE2B5B58, 0xBAEA46EF, 0xB7A96036, 0xB3687D81, 0xAD2F2D84, 0xA9EE3033, 0xA4AD16EA, 0xA06C0B5D,  //  40 [0x28 .. 0x2F]
    0xD4326D90, 0xD0F37027, 0xDDB056FE, 0xD9714B49, 0xC7361B4C, 0xC3F706FB, 0xCEB42022, 0xCA753D95,  //  48 [0x30 .. 0x37]
    0xF23A8028, 0xF6FB9D9F, 0xFBB8BB46, 0xFF79A6F1, 0xE13EF6F4, 0xE5FFEB43, 0xE8BCCD9A, 0xEC7DD02D,  //  56 [0x38 .. 0x3F]
    0x34867077, 0x30476DC0, 0x3D044B19, 0x39C556AE, 0x278206AB, 0x23431B1C, 0x2E003DC5, 0x2AC12072,  //  64 [0x40 .. 0x47]
    0x128E9DCF, 0x164F8078, 0x1B0CA6A1, 0x1FCDBB16, 0x018AEB13, 0x054BF6A4, 0x0808D07D, 0x0CC9CDCA,  //  72 [0x48 .. 0x4F]
    0x7897AB07, 0x7C56B6B0, 0x71159069, 0x75D48DDE, 0x6B93DDDB, 0x6F52C06C, 0x6211E6B5, 0x66D0FB02,  //  80 [0x50 .. 0x57]
    0x5E9F46BF, 0x5A5E5B08, 0x571D7DD1, 0x53DC6066, 0x4D9B3063, 0x495A2DD4, 0x44190B0D, 0x40D816BA,  //  88 [0x58 .. 0x5F]
    0xACA5C697, 0xA864DB20, 0xA527FDF9, 0xA1E6E04E, 0xBFA1B04B, 0xBB60ADFC, 0xB6238B25, 0xB2E29692,  //  96 [0x60 .. 0x67]
    0x8AAD2B2F, 0x8E6C3698, 0x832F1041, 0x87EE0DF6, 0x99A95DF3, 0x9D684044, 0x902B669D, 0x94EA7B2A,  // 104 [0x68 .. 0x6F]
    0xE0B41DE7, 0xE4750050, 0xE9362689, 0xEDF73B3E, 0xF3B06B3B, 0xF771768C, 0xFA325055, 0xFEF34DE2,  // 112 [0x70 .. 0x77]
    0xC6BCF05F, 0xC27DEDE8, 0xCF3ECB31, 0xCBFFD686, 0xD5B88683, 0xD1799B34, 0xDC3ABDED, 0xD8FBA05A,  // 120 [0x78 .. 0x7F]
    0x690CE0EE, 0x6DCDFD59, 0x608EDB80, 0x644FC637, 0x7A089632, 0x7EC98B85, 0x738AAD5C, 0x774BB0EB,  // 128 [0x80 .. 0x87]
    0x4F040D56, 0x4BC510E1, 0x46863638, 0x42472B8F, 0x5C007B8A, 0x58C1663D, 0x558240E4, 0x51435D53,  // 136 [0x88 .. 0x8F]
    0x251D3B9E, 0x21DC2629, 0x2C9F00F0, 0x285E1D47, 0x36194D42, 0x32D850F5, 0x3F9B762C, 0x3B5A6B9B,  // 144 [0x90 .. 0x97]
    0x0315D626, 0x07D4CB91, 0x0A97ED48, 0x0E56F0FF, 0x1011A0FA, 0x14D0BD4D, 0x19939B94, 0x1D528623,  // 152 [0x98 .. 0x9F]
    0xF12F560E, 0xF5EE4BB9, 0xF8AD6D60, 0xFC6C70D7, 0xE22B20D2, 0xE6EA3D65, 0xEBA91BBC, 0xEF68060B,  // 160 [0xA0 .. 0xA7]
    0xD727BBB6, 0xD3E6A601, 0xDEA580D8, 0xDA649D6F, 0xC423CD6A, 0xC0E2D0DD, 0xCDA1F604, 0xC960EBB3,  // 168 [0xA8 .. 0xAF]
    0xBD3E8D7E, 0xB9FF90C9, 0xB4BCB610, 0xB07DABA7, 0xAE3AFBA2, 0xAAFBE615, 0xA7B8C0CC, 0xA379DD7B,  // 176 [0xB0 .. 0xB7]
    0x9B3660C6, 0x9FF77D71, 0x92B45BA8, 0x9675461F, 0x8832161A, 0x8CF30BAD, 0x81B02D74, 0x857130C3,  // 184 [0xB8 .. 0xBF]
    0x5D8A9099, 0x594B8D2E, 0x5408ABF7, 0x50C9B640, 0x4E8EE645, 0x4A4FFBF2, 0x470CDD2B, 0x43CDC09C,  // 192 [0xC0 .. 0xC7]
    0x7B827D21, 0x7F436096, 0x7200464F, 0x76C15BF8, 0x68860BFD, 0x6C47164A, 0x61043093, 0x65C52D24,  // 200 [0xC8 .. 0xCF]
    0x119B4BE9, 0x155A565E, 0x18197087, 0x1CD86D30, 0x029F3D35, 0x065E2082, 0x0B1D065B, 0x0FDC1BEC,  // 208 [0xD0 .. 0xD7]
    0x3793A651, 0x3352BBE6, 0x3E119D3F, 0x3AD08088, 0x2497D08D, 0x2056CD3A, 0x2D15EBE3, 0x29D4F654,  // 216 [0xD8 .. 0xDF]
    0xC5A92679, 0xC1683BCE, 0xCC2B1D17, 0xC8EA00A0, 0xD6AD50A5, 0xD26C4D12, 0xDF2F6BCB, 0xDBEE767C,  // 224 [0xE0 .. 0xE7]
    0xE3A1CBC1, 0xE760D676, 0xEA23F0AF, 0xEEE2ED18, 0xF0A5BD1D, 0xF464A0AA, 0xF9278673, 0xFDE69BC4,  // 232 [0xE8 .. 0xEF]
    0x89B8FD09, 0x8D79E0BE, 0x803AC667, 0x84FBDBD0, 0x9ABC8BD5, 0x9E7D9662, 0x933EB0BB, 0x97FFAD0C,  // 240 [0xF0 .. 0xF7]
    0xAFB010B1, 0xAB710D06, 0xA6322BDF, 0xA2F33668, 0xBCB4666D, 0xB8757BDA, 0xB5365D03, 0xB1F740B4,  // 248 [0xF8 .. 0xFF]
};

#define BLOCK_MAX (65535)

#define CHARS_LEN (40)
__constant__ char CHARS[CHARS_LEN + 1] = "-.0123456789>_abcdefghijklmnopqrstuvwxyz";
//     = "-.0123456789>_abcdefghijklmnopqrstuvwxyz"
//     = "_lmnohijkdefgabcxyztuvwpqrs-.>8945670123"
//     = "_tuvwpqrsxyzdefgabclmnohijk45670123>89-."
//     = "89>01234567-._xyzpqrstuvwhijklmnoabcdefg"
//     = "-.1032547698>_acbedgfihkjmlonqpsrutwvyxz"
//     = "_mlonihkjedgfacbyxzutwvqpsr-.>9854761032"
//     = "_utwvqpsryxzedgfacbmlonihkj54761032>98-."
//     = "98>10325476-._yxzqpsrutwvihkjmlonacbedgf"
//     = "_nolmjkhifgdebcazxyvwturspq.->8967452301"
//     = ".-2301674589>_bcafgdejkhinolmrspqvwtuzxy"
//     = "89>23016745.-_zxyrspqvwtujkhinolmbcafgde"
//     = "_vwturspqzxyfgdebcanolmjkhi67452301>89.-"
//     = "_onmlkjihgfedcbazyxwvutsrqp.->9876543210"
//     = ".-3210765498>_cbagfedkjihonmlsrqpwvutzyx"
//     = "98>32107654.-_zyxsrqpwvutkjihonmlcbagfed"
//     = "_wvutsrqpzyxgfedcbaonmlkjih76543210>98.-"
//     = ">8945670123-._xyztuvwpqrslmnohijkdefgabc"
//     = "_pqrstuvwxyzabcdefghijklmno0123456789>-."
//     = "_hijklmnoabcdefgxyzpqrstuvw-.89>01234567"
//     = "-.45670123>89_defgabclmnohijktuvwpqrsxyz"
//     = ">9854761032-._yxzutwvqpsrmlonihkjedgfacb"
//     = "_qpsrutwvyxzacbedgfihkjmlon1032547698>-."
//     = "_ihkjmlonacbedgfyxzqpsrutwv-.98>10325476"
//     = "-.54761032>98_edgfacbmlonihkjutwvqpsryxz"
//     = "_rspqvwtuzxybcafgdejkhinolm2301674589>.-"
//     = ">8967452301.-_zxyvwturspqnolmjkhifgdebca"
//     = ".-67452301>89_fgdebcanolmjkhivwturspqzxy"
//     = "_jkhinolmbcafgdezxyrspqvwtu.-89>23016745"
//     = "_srqpwvutzyxcbagfedkjihonml3210765498>.-"
//     = ">9876543210.-_zyxwvutsrqponmlkjihgfedcba"
//     = ".-76543210>98_gfedcbaonmlkjihwvutsrqpzyx"
//     = "_kjihonmlcbagfedzyxsrqpwvut.-98>32107654"
//     = "_yxzutwvqpsrmlonihkjedgfacb>9854761032-."
//     = "1032547698>-._qpsrutwvyxzacbedgfihkjmlon"
//     = "-.98>10325476_ihkjmlonacbedgfyxzqpsrutwv"
//     = "_edgfacbmlonihkjutwvqpsryxz-.54761032>98"
//     = "_xyztuvwpqrslmnohijkdefgabc>8945670123-."
//     = "0123456789>-._pqrstuvwxyzabcdefghijklmno"
//     = "-.89>01234567_hijklmnoabcdefgxyzpqrstuvw"
//     = "_defgabclmnohijktuvwpqrsxyz-.45670123>89"
//     = "3210765498>.-_srqpwvutzyxcbagfedkjihonml"
//     = "_zyxwvutsrqponmlkjihgfedcba>9876543210.-"
//     = "_gfedcbaonmlkjihwvutsrqpzyx.-76543210>98"
//     = ".-98>32107654_kjihonmlcbagfedzyxsrqpwvut"
//     = "2301674589>.-_rspqvwtuzxybcafgdejkhinolm"
//     = "_zxyvwturspqnolmjkhifgdebca>8967452301.-"
//     = "_fgdebcanolmjkhivwturspqzxy.-67452301>89"
//     = ".-89>23016745_jkhinolmbcafgdezxyrspqvwtu"
//     = "_acbedgfihkjmlonqpsrutwvyxz-.1032547698>"
//     = "-.>9854761032_mlonihkjedgfacbyxzutwvqpsr"
//     = "54761032>98-._utwvqpsryxzedgfacbmlonihkj"
//     = "_yxzqpsrutwvihkjmlonacbedgf98>10325476-."
//     = "_abcdefghijklmnopqrstuvwxyz-.0123456789>"
//     = "-.>8945670123_lmnohijkdefgabcxyztuvwpqrs"
//     = "45670123>89-._tuvwpqrsxyzdefgabclmnohijk"
//     = "_xyzpqrstuvwhijklmnoabcdefg89>01234567-."
//     = ".->9876543210_onmlkjihgfedcbazyxwvutsrqp"
//     = "_cbagfedkjihonmlsrqpwvutzyx.-3210765498>"
//     = "_zyxsrqpwvutkjihonmlcbagfed98>32107654.-"
//     = "76543210>98.-_wvutsrqpzyxgfedcbaonmlkjih"
//     = ".->8967452301_nolmjkhifgdebcazxyvwturspq"
//     = "_bcafgdejkhinolmrspqvwtuzxy.-2301674589>"
//     = "_zxyrspqvwtujkhinolmbcafgde89>23016745.-"
//     = "67452301>89.-_vwturspqzxyfgdebcanolmjkhi"
//     = "_zyxwvutsrqponmlkjihgfedcba>9876543210.-"
//     = "3210765498>.-_srqpwvutzyxcbagfedkjihonml"
//     = ".-98>32107654_kjihonmlcbagfedzyxsrqpwvut"
//     = "_gfedcbaonmlkjihwvutsrqpzyx.-76543210>98"
//     = "_zxyvwturspqnolmjkhifgdebca>8967452301.-"
//     = "2301674589>.-_rspqvwtuzxybcafgdejkhinolm"
//     = ".-89>23016745_jkhinolmbcafgdezxyrspqvwtu"
//     = "_fgdebcanolmjkhivwturspqzxy.-67452301>89"
//     = "1032547698>-._qpsrutwvyxzacbedgfihkjmlon"
//     = "_yxzutwvqpsrmlonihkjedgfacb>9854761032-."
//     = "_edgfacbmlonihkjutwvqpsryxz-.54761032>98"
//     = "-.98>10325476_ihkjmlonacbedgfyxzqpsrutwv"
//     = "0123456789>-._pqrstuvwxyzabcdefghijklmno"
//     = "_xyztuvwpqrslmnohijkdefgabc>8945670123-."
//     = "_defgabclmnohijktuvwpqrsxyz-.45670123>89"
//     = "-.89>01234567_hijklmnoabcdefgxyzpqrstuvw"
//     = "_cbagfedkjihonmlsrqpwvutzyx.-3210765498>"
//     = ".->9876543210_onmlkjihgfedcbazyxwvutsrqp"
//     = "76543210>98.-_wvutsrqpzyxgfedcbaonmlkjih"
//     = "_zyxsrqpwvutkjihonmlcbagfed98>32107654.-"
//     = "_bcafgdejkhinolmrspqvwtuzxy.-2301674589>"
//     = ".->8967452301_nolmjkhifgdebcazxyvwturspq"
//     = "67452301>89.-_vwturspqzxyfgdebcanolmjkhi"
//     = "_zxyrspqvwtujkhinolmbcafgde89>23016745.-"
//     = "-.>9854761032_mlonihkjedgfacbyxzutwvqpsr"
//     = "_acbedgfihkjmlonqpsrutwvyxz-.1032547698>"
//     = "_yxzqpsrutwvihkjmlonacbedgf98>10325476-."
//     = "54761032>98-._utwvqpsryxzedgfacbmlonihkj"
//     = "-.>8945670123_lmnohijkdefgabcxyztuvwpqrs"
//     = "_abcdefghijklmnopqrstuvwxyz-.0123456789>"
//     = "_xyzpqrstuvwhijklmnoabcdefg89>01234567-."
//     = "45670123>89-._tuvwpqrsxyzdefgabclmnohijk"
//     = ".-2301674589>_bcafgdejkhinolmrspqvwtuzxy"
//     = "_nolmjkhifgdebcazxyvwturspq.->8967452301"
//     = "_vwturspqzxyfgdebcanolmjkhi67452301>89.-"
//     = "89>23016745.-_zxyrspqvwtujkhinolmbcafgde"
//     = ".-3210765498>_cbagfedkjihonmlsrqpwvutzyx"
//     = "_onmlkjihgfedcbazyxwvutsrqp.->9876543210"
//     = "_wvutsrqpzyxgfedcbaonmlkjih76543210>98.-"
//     = "98>32107654.-_zyxsrqpwvutkjihonmlcbagfed"
//     = "_lmnohijkdefgabcxyztuvwpqrs-.>8945670123"
//     = "-.0123456789>_abcdefghijklmnopqrstuvwxyz"
//     = "89>01234567-._xyzpqrstuvwhijklmnoabcdefg"
//     = "_tuvwpqrsxyzdefgabclmnohijk45670123>89-."
//     = "_mlonihkjedgfacbyxzutwvqpsr-.>9854761032"
//     = "-.1032547698>_acbedgfihkjmlonqpsrutwvyxz"
//     = "98>10325476-._yxzqpsrutwvihkjmlonacbedgf"
//     = "_utwvqpsryxzedgfacbmlonihkj54761032>98-."
//     = ">8967452301.-_zxyvwturspqnolmjkhifgdebca"
//     = "_rspqvwtuzxybcafgdejkhinolm2301674589>.-"
//     = "_jkhinolmbcafgdezxyrspqvwtu.-89>23016745"
//     = ".-67452301>89_fgdebcanolmjkhivwturspqzxy"
//     = ">9876543210.-_zyxwvutsrqponmlkjihgfedcba"
//     = "_srqpwvutzyxcbagfedkjihonml3210765498>.-"
//     = "_kjihonmlcbagfedzyxsrqpwvut.-98>32107654"
//     = ".-76543210>98_gfedcbaonmlkjihwvutsrqpzyx"
//     = "_pqrstuvwxyzabcdefghijklmno0123456789>-."
//     = ">8945670123-._xyztuvwpqrslmnohijkdefgabc"
//     = "-.45670123>89_defgabclmnohijktuvwpqrsxyz"
//     = "_hijklmnoabcdefgxyzpqrstuvw-.89>01234567"
//     = "_qpsrutwvyxzacbedgfihkjmlon1032547698>-."
//     = ">9854761032-._yxzutwvqpsrmlonihkjedgfacb"
//     = "-.54761032>98_edgfacbmlonihkjutwvqpsryxz"
//     = "_ihkjmlonacbedgfyxzqpsrutwv-.98>10325476"
//     = "rspqvwtuzxybcafgdejkhinolm_2301674589>.-"
//     = ">8967452301.-zxyvwturspqnolmjkhifgdebca_"
//     = ".-67452301>89fgdebcanolmjkhivwturspqzxy_"
//     = "jkhinolmbcafgdezxyrspqvwtu_.-89>23016745"
//     = "srqpwvutzyxcbagfedkjihonml_3210765498>.-"
//     = ">9876543210.-zyxwvutsrqponmlkjihgfedcba_"
//     = ".-76543210>98gfedcbaonmlkjihwvutsrqpzyx_"
//     = "kjihonmlcbagfedzyxsrqpwvut_.-98>32107654"
//     = ">8945670123-.xyztuvwpqrslmnohijkdefgabc_"
//     = "pqrstuvwxyzabcdefghijklmno_0123456789>-."
//     = "hijklmnoabcdefgxyzpqrstuvw_-.89>01234567"
//     = "-.45670123>89defgabclmnohijktuvwpqrsxyz_"
//     = ">9854761032-.yxzutwvqpsrmlonihkjedgfacb_"
//     = "qpsrutwvyxzacbedgfihkjmlon_1032547698>-."
//     = "ihkjmlonacbedgfyxzqpsrutwv_-.98>10325476"
//     = "-.54761032>98edgfacbmlonihkjutwvqpsryxz_"
//     = "nolmjkhifgdebcazxyvwturspq_.->8967452301"
//     = ".-2301674589>bcafgdejkhinolmrspqvwtuzxy_"
//     = "89>23016745.-zxyrspqvwtujkhinolmbcafgde_"
//     = "vwturspqzxyfgdebcanolmjkhi_67452301>89.-"
//     = "onmlkjihgfedcbazyxwvutsrqp_.->9876543210"
//     = ".-3210765498>cbagfedkjihonmlsrqpwvutzyx_"
//     = "98>32107654.-zyxsrqpwvutkjihonmlcbagfed_"
//     = "wvutsrqpzyxgfedcbaonmlkjih_76543210>98.-"
//     = "-.0123456789>abcdefghijklmnopqrstuvwxyz_"
//     = "lmnohijkdefgabcxyztuvwpqrs_-.>8945670123"
//     = "tuvwpqrsxyzdefgabclmnohijk_45670123>89-."
//     = "89>01234567-.xyzpqrstuvwhijklmnoabcdefg_"
//     = "-.1032547698>acbedgfihkjmlonqpsrutwvyxz_"
//     = "mlonihkjedgfacbyxzutwvqpsr_-.>9854761032"
//     = "utwvqpsryxzedgfacbmlonihkj_54761032>98-."
//     = "98>10325476-.yxzqpsrutwvihkjmlonacbedgf_"
//     = ".->9876543210onmlkjihgfedcbazyxwvutsrqp_"
//     = "cbagfedkjihonmlsrqpwvutzyx_.-3210765498>"
//     = "zyxsrqpwvutkjihonmlcbagfed_98>32107654.-"
//     = "76543210>98.-wvutsrqpzyxgfedcbaonmlkjih_"
//     = ".->8967452301nolmjkhifgdebcazxyvwturspq_"
//     = "bcafgdejkhinolmrspqvwtuzxy_.-2301674589>"
//     = "zxyrspqvwtujkhinolmbcafgde_89>23016745.-"
//     = "67452301>89.-vwturspqzxyfgdebcanolmjkhi_"
//     = "acbedgfihkjmlonqpsrutwvyxz_-.1032547698>"
//     = "-.>9854761032mlonihkjedgfacbyxzutwvqpsr_"
//     = "54761032>98-.utwvqpsryxzedgfacbmlonihkj_"
//     = "yxzqpsrutwvihkjmlonacbedgf_98>10325476-."
//     = "abcdefghijklmnopqrstuvwxyz_-.0123456789>"
//     = "-.>8945670123lmnohijkdefgabcxyztuvwpqrs_"
//     = "45670123>89-.tuvwpqrsxyzdefgabclmnohijk_"
//     = "xyzpqrstuvwhijklmnoabcdefg_89>01234567-."
//     = "3210765498>.-srqpwvutzyxcbagfedkjihonml_"
//     = "zyxwvutsrqponmlkjihgfedcba_>9876543210.-"
//     = "gfedcbaonmlkjihwvutsrqpzyx_.-76543210>98"
//     = ".-98>32107654kjihonmlcbagfedzyxsrqpwvut_"
//     = "2301674589>.-rspqvwtuzxybcafgdejkhinolm_"
//     = "zxyvwturspqnolmjkhifgdebca_>8967452301.-"
//     = "fgdebcanolmjkhivwturspqzxy_.-67452301>89"
//     = ".-89>23016745jkhinolmbcafgdezxyrspqvwtu_"
//     = "yxzutwvqpsrmlonihkjedgfacb_>9854761032-."
//     = "1032547698>-.qpsrutwvyxzacbedgfihkjmlon_"
//     = "-.98>10325476ihkjmlonacbedgfyxzqpsrutwv_"
//     = "edgfacbmlonihkjutwvqpsryxz_-.54761032>98"
//     = "xyztuvwpqrslmnohijkdefgabc_>8945670123-."
//     = "0123456789>-.pqrstuvwxyzabcdefghijklmno_"
//     = "-.89>01234567hijklmnoabcdefgxyzpqrstuvw_"
//     = "defgabclmnohijktuvwpqrsxyz_-.45670123>89"
//     = "-.>9854761032mlonihkjedgfacbyxzutwvqpsr_"
//     = "acbedgfihkjmlonqpsrutwvyxz_-.1032547698>"
//     = "yxzqpsrutwvihkjmlonacbedgf_98>10325476-."
//     = "54761032>98-.utwvqpsryxzedgfacbmlonihkj_"
//     = "-.>8945670123lmnohijkdefgabcxyztuvwpqrs_"
//     = "abcdefghijklmnopqrstuvwxyz_-.0123456789>"
//     = "xyzpqrstuvwhijklmnoabcdefg_89>01234567-."
//     = "45670123>89-.tuvwpqrsxyzdefgabclmnohijk_"
//     = "cbagfedkjihonmlsrqpwvutzyx_.-3210765498>"
//     = ".->9876543210onmlkjihgfedcbazyxwvutsrqp_"
//     = "76543210>98.-wvutsrqpzyxgfedcbaonmlkjih_"
//     = "zyxsrqpwvutkjihonmlcbagfed_98>32107654.-"
//     = "bcafgdejkhinolmrspqvwtuzxy_.-2301674589>"
//     = ".->8967452301nolmjkhifgdebcazxyvwturspq_"
//     = "67452301>89.-vwturspqzxyfgdebcanolmjkhi_"
//     = "zxyrspqvwtujkhinolmbcafgde_89>23016745.-"
//     = "1032547698>-.qpsrutwvyxzacbedgfihkjmlon_"
//     = "yxzutwvqpsrmlonihkjedgfacb_>9854761032-."
//     = "edgfacbmlonihkjutwvqpsryxz_-.54761032>98"
//     = "-.98>10325476ihkjmlonacbedgfyxzqpsrutwv_"
//     = "0123456789>-.pqrstuvwxyzabcdefghijklmno_"
//     = "xyztuvwpqrslmnohijkdefgabc_>8945670123-."
//     = "defgabclmnohijktuvwpqrsxyz_-.45670123>89"
//     = "-.89>01234567hijklmnoabcdefgxyzpqrstuvw_"
//     = "zyxwvutsrqponmlkjihgfedcba_>9876543210.-"
//     = "3210765498>.-srqpwvutzyxcbagfedkjihonml_"
//     = ".-98>32107654kjihonmlcbagfedzyxsrqpwvut_"
//     = "gfedcbaonmlkjihwvutsrqpzyx_.-76543210>98"
//     = "zxyvwturspqnolmjkhifgdebca_>8967452301.-"
//     = "2301674589>.-rspqvwtuzxybcafgdejkhinolm_"
//     = ".-89>23016745jkhinolmbcafgdezxyrspqvwtu_"
//     = "fgdebcanolmjkhivwturspqzxy_.-67452301>89"
//     = "pqrstuvwxyzabcdefghijklmno_0123456789>-."
//     = ">8945670123-.xyztuvwpqrslmnohijkdefgabc_"
//     = "-.45670123>89defgabclmnohijktuvwpqrsxyz_"
//     = "hijklmnoabcdefgxyzpqrstuvw_-.89>01234567"
//     = "qpsrutwvyxzacbedgfihkjmlon_1032547698>-."
//     = ">9854761032-.yxzutwvqpsrmlonihkjedgfacb_"
//     = "-.54761032>98edgfacbmlonihkjutwvqpsryxz_"
//     = "ihkjmlonacbedgfyxzqpsrutwv_-.98>10325476"
//     = ">8967452301.-zxyvwturspqnolmjkhifgdebca_"
//     = "rspqvwtuzxybcafgdejkhinolm_2301674589>.-"
//     = "jkhinolmbcafgdezxyrspqvwtu_.-89>23016745"
//     = ".-67452301>89fgdebcanolmjkhivwturspqzxy_"
//     = ">9876543210.-zyxwvutsrqponmlkjihgfedcba_"
//     = "srqpwvutzyxcbagfedkjihonml_3210765498>.-"
//     = "kjihonmlcbagfedzyxsrqpwvut_.-98>32107654"
//     = ".-76543210>98gfedcbaonmlkjihwvutsrqpzyx_"
//     = "lmnohijkdefgabcxyztuvwpqrs_-.>8945670123"
//     = "-.0123456789>abcdefghijklmnopqrstuvwxyz_"
//     = "89>01234567-.xyzpqrstuvwhijklmnoabcdefg_"
//     = "tuvwpqrsxyzdefgabclmnohijk_45670123>89-."
//     = "mlonihkjedgfacbyxzutwvqpsr_-.>9854761032"
//     = "-.1032547698>acbedgfihkjmlonqpsrutwvyxz_"
//     = "98>10325476-.yxzqpsrutwvihkjmlonacbedgf_"
//     = "utwvqpsryxzedgfacbmlonihkj_54761032>98-."
//     = ".-2301674589>bcafgdejkhinolmrspqvwtuzxy_"
//     = "nolmjkhifgdebcazxyvwturspq_.->8967452301"
//     = "vwturspqzxyfgdebcanolmjkhi_67452301>89.-"
//     = "89>23016745.-zxyrspqvwtujkhinolmbcafgde_"
//     = ".-3210765498>cbagfedkjihonmlsrqpwvutzyx_"
//     = "onmlkjihgfedcbazyxwvutsrqp_.->9876543210"
//     = "wvutsrqpzyxgfedcbaonmlkjih_76543210>98.-"
//     = "98>32107654.-zyxsrqpwvutkjihonmlcbagfed_"

__global__
void kernel(std::uint32_t hashOffset)
{
    crc32_t hash = blockIdx.x + hashOffset;
    crc32_t newHash = (hash >> 8) ^ crc32_table[(threadIdx.x ^ hash) & 0xff];

    //for (int i = 0; i < 1048576; ++i)
    //{
    //    std::uint32_t X = ((16 * (268435456 / 256)) * i);
    //    if (newHash == X)
    //    {
    //        printf("%d %u %u %c\n", i, hash, newHash, CHARS[threadIdx.x]);
    //    }
    //}
    std::uint32_t X = ((16 * (268435456 / 256)));
    if (newHash % X == 0)
    {
        printf("%u %u %u\n", newHash / X, hash, threadIdx.x);
    }
}

int main()
{
    for (std::uint32_t i = 0; i < CRC32_MAX / BLOCK_MAX; ++i)
    {
        kernel<<<BLOCK_MAX, 256>>>(i * BLOCK_MAX);
    }
    CUDA_ASSERT(hipPeekAtLastError());
    CUDA_ASSERT(hipDeviceSynchronize());
    hipDeviceReset();
	return 0;
}
